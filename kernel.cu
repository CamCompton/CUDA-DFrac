#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>
#include <GL/glew.h>



GLfloat* D_cordinatePos;
GLfloat* D_colors;
unsigned int* D_indices;
unsigned int* D_indicesNew;
hipStream_t iterateStream, indexStream;

//Calculates the indices for a the next iteration of the fratal
extern "C" __global__ void indicesKernel(unsigned int* indices, unsigned int* indicesNew, unsigned int size){
  int i = blockDim.x * blockIdx.x+threadIdx.x;
  if(i < size * 2 -1){
    if((i%2) == 0){
      indicesNew[i] = indices[i/2];
    }
    else{
      indicesNew[i] = size + i/2;
    }
  }
}
//Copies the newest indices to the area of devices memory that is used by OpenGL, much faster than MemCpy
extern "C" __global__ void indicesCopyKernel(unsigned int* deviceIndicesPtr,unsigned int* indices, unsigned int* indicesNew, unsigned int size){
  int i = blockDim.x * blockIdx.x+threadIdx.x;

  if(i < size){
    deviceIndicesPtr[i] = indicesNew[i];
  }
}


//almost the same as serial code, includes some control flow optimizations
extern "C" __global__ void iterateKernel(GLfloat* deviceVertPtr,GLfloat* deviceColorPtr,unsigned int* indices,const unsigned int size){
  int i = blockDim.x * blockIdx.x+threadIdx.x;

  if(i < size-1){
  GLfloat startX = deviceVertPtr[indices[i]*2];
  GLfloat startY = deviceVertPtr[(indices[i]*2)+1];

  GLfloat nextX = deviceVertPtr[indices[i+1] * 2];
  GLfloat nextY = deviceVertPtr[indices[i+1] * 2 + 1];

  GLfloat newStartX;
  GLfloat newStartY;


    if(startY == nextY){
      if( (nextX > startX && (i%2) == 0) || (nextX <= startX && (i%2) == 1)){
        //cp 1
        newStartX = (startX + nextX)/2;
        newStartY = startY + abs(nextX - startX)/2;
      }
      else{
        //cp 2
        newStartX = (startX + nextX)/2;
        newStartY = startY - abs(nextX - startX)/2;
      }
    }
    else if(startX == nextX){
      if((nextY > startY && (i%2) == 0) || (nextY <= startY && (i%2) == 1)){
        //cp 3
        newStartX = startX - abs(nextY - startY)/2;
        newStartY = (nextY + startY)/2;
      }
      else{
        //cp 4
        newStartX = startX + abs(nextY - startY)/2;
        newStartY = (nextY + startY)/2;
      }
    }
    else{
      if(( ( (nextX > startX && nextY > startY) || (nextX < startX && nextY < startY)) && (i%2) == 0)
        || (!((nextX > startX && nextY > startY) || (nextX < startX && nextY < startY)) && (i%2) == 1)){
        //cp 5
        newStartX = startX;
        newStartY = nextY;
      }
      else{
        //cp 6
        newStartX = nextX;
        newStartY = startY;
      }
    }
    	deviceVertPtr[(size+i)*2] = newStartX;
    	deviceVertPtr[(size+i)*2 +1] = newStartY;

    	float inc = 2.0/size ;

    	deviceColorPtr[(size+i)*3] = 2.0-(inc*i);
    	deviceColorPtr[(size+i)*3+1] = inc*i;
    	deviceColorPtr[(size+i)*3+2] = 0;

    }



  }


//kernel launcher function does not hipMemcpy at all.
//this function is called repeatedly.
//color and cordinate data is automaticly stored in device memory locations used by OpenGL.
// VBO index array must be copeied to an OpeGL location seperatly because the operation cannot be done in place easily.
extern "C" int kernelLauncher(GLfloat* deviceVertPtr,GLfloat* deviceColorPtr,unsigned int* deviceIndicesPtr,unsigned int size) {
  unsigned int* temp;
  temp = D_indices;
  D_indices = D_indicesNew;
  D_indicesNew = temp;

  int blockSize = 1024;
  int gridSize = ((size-1)/1024)+1;
  iterateKernel<<<gridSize, blockSize,0,iterateStream>>>(deviceVertPtr,deviceColorPtr,D_indices,size);
  gridSize = ((size*2-1)/1024)+1;
  indicesKernel<<<gridSize, blockSize,0,indexStream>>>(D_indices,D_indicesNew,size);

  hipDeviceSynchronize();

  return 0;
}

// kernel init functon mallocs and initializes indicies memory location.
// not needed for color and position data because OpenGL buffers and cuda register buffers does this for us.
void kernelInit(unsigned int* H_indices,unsigned int size, unsigned int totalSize) {
  hipStreamCreate(&iterateStream);
  hipStreamCreate(&indexStream);

  hipMalloc(&D_indices,      totalSize*sizeof(unsigned int));
  hipMalloc(&D_indicesNew,   totalSize*sizeof(unsigned int));

  hipMemcpy(D_indicesNew,     H_indices,      size * sizeof(unsigned int), hipMemcpyHostToDevice);
}
//called only once
//copies over the indices array and frees memory. 
void kernelClose(unsigned int* deviceIndicesPtr,unsigned int* H_indices,unsigned int size){

  int blockSize = 1024;
  int gridSize = ((size*2-1)/1024)+1;
  indicesCopyKernel<<<gridSize, blockSize,0,indexStream>>>(deviceIndicesPtr,D_indices,D_indicesNew,size);
  hipDeviceSynchronize();

  hipFree(D_indices);
  hipFree(D_indicesNew);


}
